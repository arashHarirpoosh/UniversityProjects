#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include ""
#include<stdlib.h>
#include <stdio.h>

struct threadInfo
{
	int globalThread;
	int block;
	int warp;
	int thread;
};

hipError_t printWithCuda(threadInfo *t, unsigned int numOfBlock, unsigned int block_size);
void printThreadInfo(threadInfo * t, int size);

__global__ void printKernel(threadInfo *t)
{
	int globalThread = threadIdx.x + blockIdx.x * blockDim.x;
	int warp = threadIdx.x / warpSize;
	t[globalThread] = { globalThread, blockIdx.x, warp, threadIdx.x };

}

int main()
{
	const int numOfBlock = 2;
	const int block_size = 64;
	int size = numOfBlock * block_size;
	hipEvent_t start;
	hipEventCreate(&start);
	hipEvent_t stop;
	hipEventCreate(&stop);

	threadInfo *t = (threadInfo*)malloc(sizeof(threadInfo) * size);

	hipEventRecord(start, NULL);

	// Print vectors in parallel.
	hipError_t cudaStatus = printWithCuda(t, numOfBlock, block_size);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	// Print Threads info
	printThreadInfo(t, size);

	hipEventRecord(stop, NULL);

	cudaStatus = hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	cudaStatus = hipEventElapsedTime(&msecTotal, start, stop);

	fprintf(stderr, "Elapsed Time is %f ms \n", msecTotal);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	return 0;
}

void printThreadInfo(threadInfo * t, int size) {
	for (int i = 0; i < size; i++)
	{
		printf("Calculated Thread: %d - Block: %d - Warp: %d - Thread: %d \n", t[i].globalThread, t[i].block, t[i].warp, t[i].thread);
	}

}

// Helper function for using CUDA to add vectors in parallel.
hipError_t printWithCuda(threadInfo *t, unsigned int numOfBlock, unsigned int block_size)
{
	threadInfo *dev_t;
	hipError_t cudaStatus;
	const int size = numOfBlock * block_size;
	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_t, size * sizeof(threadInfo));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Launch a kernel on the GPU with one thread for each element.
	printKernel << < numOfBlock, block_size >> > (dev_t);

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}


	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(t, dev_t, size * sizeof(threadInfo), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_t);

	return cudaStatus;
}
