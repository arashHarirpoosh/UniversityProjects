#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include <hip/hip_runtime.h>

hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size);
void fillMat(int * v, int matSizeX, int matSizeY);
void printMat(int * v, int matSizeX, int matSizeY);

__global__ void addKernel(int *c, const int *a, const int *b, int size)
{
    int i = threadIdx.x + blockDim.x * blockIdx.x;

	if (i < size) {
		c[i] = a[i] + b[i];
	}
}


int main()
{
	const int matSizeX = 1000;
	const int matSizeY = 10000;
	const int arraySize = matSizeX*matSizeY;

	int * a;
	int * b;
	int * c;
	;
	a = (int*)malloc(sizeof(int)*matSizeX*matSizeY);
	b = (int*)malloc(sizeof(int)*matSizeX*matSizeY);
	c = (int*)malloc(sizeof(int)*matSizeX*matSizeY);
	fillMat(a, matSizeX, matSizeY);
	fillMat(b, matSizeX, matSizeY);
    // Add vectors in parallel.
    hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

	/*printMat(a, matSizeX, matSizeY);
	printMat(b, matSizeX, matSizeY);
	printMat(c, matSizeX, matSizeY);*/

    // hipDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

void fillMat(int * v, int matSizeX, int matSizeY) {
	static int L = 0;
	for (int i = 0; i < matSizeX; i++) {
		for (int j = 0; j < matSizeY; j++)
			v[i*matSizeY + j] = L++;
	}
}
void printMat(int * v, int matSizeX, int matSizeY) {
	int i;
	printf("[-] Vector elements: \n");
	for (int i = 0; i < matSizeX; i++) {
		for (int j = 0; j < matSizeY; j++)
			printf("%d ", v[i*matSizeY + j]);
		printf("\n");
	}
	printf("\b\b \n");
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int *c, const int *a, const int *b, unsigned int size)
{
    int *dev_a = 0;
    int *dev_b = 0;
    int *dev_c = 0;
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
	//checkError(cudaStatus, 0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
        goto Error;
    }

    // Allocate GPU buffers for three vectors (two input, one output)    .
    cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	//checkError(cudaStatus, 1);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMalloc failed!");
        goto Error;
    }

    // Copy input vectors from host memory to GPU buffers.
    cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

    cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

	hipEvent_t start;
	hipEventCreate(&start);
	hipEvent_t stop;
	hipEventCreate(&stop);

	hipEventRecord(start, NULL);
	int numOfThread = 1024;
	int n = int((size - 1) / numOfThread) + 1;
    // Launch a kernel on the GPU with one thread for each element.
    addKernel<<<n, numOfThread>>>(dev_c, dev_a, dev_b, size);

	hipEventRecord(stop, NULL);
	cudaStatus = hipEventSynchronize(stop);
	float msecTotal = 0.0f;
	cudaStatus = hipEventElapsedTime(&msecTotal, start, stop);
	fprintf(stderr, "Elapsed Time is %f ms \n", msecTotal);

    // Check for any errors launching the kernel
    cudaStatus = hipGetLastError();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
        goto Error;
    }
    
    // hipDeviceSynchronize waits for the kernel to finish, and returns
    // any errors encountered during the launch.
    cudaStatus = hipDeviceSynchronize();

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
        goto Error;
    }

    // Copy output vector from GPU buffer to host memory.
    cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);

    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipMemcpy failed!");
        goto Error;
    }

Error:
    hipFree(dev_c);
    hipFree(dev_a);
    hipFree(dev_b);
    
    return cudaStatus;
}
