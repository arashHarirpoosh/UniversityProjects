
#include "hip/hip_runtime.h"


#include <stdio.h>

hipError_t printWithCuda(int blockNum, int threadNum);

__global__ void printKernel()
{
	printf("Hello CUDA Im a thread from grid %d and block %d \n", threadIdx.x, blockIdx.x);
}

int main()
{

    // print vectors in parallel.
    hipError_t cudaStatus = printWithCuda(2, 7);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addWithCuda failed!");
        return 1;
    }

    // cudaDeviceReset must be called before exiting in order for profiling and
    // tracing tools such as Nsight and Visual Profiler to show complete traces.
    cudaStatus = hipDeviceReset();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipDeviceReset failed!");
        return 1;
    }

    return 0;
}

// Helper function for using CUDA to print vectors in parallel.
hipError_t printWithCuda(int blockNum, int threadNum)
{
    hipError_t cudaStatus;

    // Choose which GPU to run on, change this on a multi-GPU system.
    cudaStatus = hipSetDevice(0);
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
    }


    // Launch a kernel on the GPU with one thread for each element.
    printKernel<<<blockNum, threadNum>>>();

	// cudaDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
	}

	// Check for any errors launching the kernel
    cudaStatus = hipGetLastError();
    if (cudaStatus != hipSuccess) {
        fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
    }
    
    return cudaStatus;
}
