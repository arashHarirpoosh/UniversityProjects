#include "hip/hip_runtime.h"

#include "hip/hip_runtime.h"
#include ""

#include <stdio.h>
#include<stdlib.h>
#include <omp.h>
#include<iostream>
#define MAX_HISTORGRAM_NUMBER 10000
#define ARRAY_SIZE 102400000

#define CHUNK_SIZE 100
#define THREAD_COUNT 1024
#define SCALER 20
#define NumOfStreams  4
hipError_t histogramWithCuda(int *a, unsigned long long int *c);

__global__ void histogramKernelSingle(unsigned long long int *c, int *a)
{
	unsigned long long int worker = blockIdx.x*blockDim.x + threadIdx.x;
	unsigned long long int start = worker * CHUNK_SIZE;
	unsigned long long int end = start + CHUNK_SIZE;
	for (int ex = 0; ex < SCALER/NumOfStreams; ex++)
		for (long long int i = start; i < end; i++)
		{
			if (i < ARRAY_SIZE)
				atomicAdd(&c[a[i]], 1);
			else
			{

				break;
			}
		}

}
int main()
{
	int* a;
	hipError_t cudaStatus;
	cudaStatus = hipHostAlloc((void**)&a, sizeof(int)*ARRAY_SIZE, hipHostMallocWriteCombined |
		hipHostMallocMapped);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipHostAlloc failed!");
		return 1;
	}
	unsigned long long int* c;
	cudaStatus = hipHostAlloc((void**)&c, sizeof(unsigned long long int)*MAX_HISTORGRAM_NUMBER, hipHostMallocWriteCombined |
		hipHostMallocMapped);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipHostAlloc failed!");
		return 1;
	}
	for (unsigned long long i = 0; i < ARRAY_SIZE; i++)
		a[i] = rand() % MAX_HISTORGRAM_NUMBER;
	for (unsigned long long i = 0; i < MAX_HISTORGRAM_NUMBER; i++)
		c[i] = 0;

	// Add vectors in parallel.
	double start_time = omp_get_wtime();
	cudaStatus = histogramWithCuda(a, c);
	double end_time = omp_get_wtime();
	std::cout << end_time - start_time;
	// = 
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	unsigned long long int* copy_c = (unsigned long long int*)malloc(sizeof(unsigned long long int)*MAX_HISTORGRAM_NUMBER);
	for (unsigned long long i = 0; i < MAX_HISTORGRAM_NUMBER; i++)
		copy_c[i] = c[i];
	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	unsigned long long int R = 0;
	for (int i = 0; i < MAX_HISTORGRAM_NUMBER; i++)
	{
		R += copy_c[i];
		//		printf("%d	", c[i]);
	}
	printf("\nCORRECT:%ld	", R / (SCALER));

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t histogramWithCuda(int *a, unsigned long long int *c)
{
	int *dev_a = 0;
	unsigned long long int *dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, ARRAY_SIZE * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}


	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, ARRAY_SIZE * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}
	// Launch a kernel on the GPU with one thread for each element.
	//// BLOCK CALCULATOR HERE


	////BLOCK CALCULATOR HERE
	int numBlock = int(ARRAY_SIZE / (THREAD_COUNT*CHUNK_SIZE));
	hipStream_t streams[NumOfStreams];
	for (int i = 0; i < NumOfStreams; i++) {
		hipStreamCreate(&streams[i]);
		// launch one worker kernel per stream
		histogramKernelSingle << <numBlock, THREAD_COUNT, 0, streams[i] >> > (dev_c, dev_a);
	}
	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, MAX_HISTORGRAM_NUMBER * sizeof(unsigned long long int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	return cudaStatus;
}
